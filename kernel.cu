#include "hip/hip_runtime.h"
#define CRT_SECURE_NO_WARNINGS
#include <iostream>
#include <vector>
#include "hip/hip_runtime.h"
#include ""
#include "lodepng.h"

__global__ void ConvolutionKernel(
	const float* devicePaddedImage,
	const unsigned int paddedWidth,
	const unsigned int paddedHeight,
	const float* filter, const int S,
	float* result, const unsigned int width, const unsigned int height)
{
	// Postavlja se veli�ina filtera na osnovu veli�ine paddinga
	unsigned int paddingSize = S;
	unsigned int filterSize = 2 * S + 1;
	// Varijabla sum se koristi prilikom normaliziranja ra�una.
	unsigned int sum = 0;

	// Ra�unaju se koordinate trenutnog pixela preko infromacije sadr�ane u bloku tj. threadu koji ga obra�uje
	// Potrebno je i dodati paddingSize jer GPU radi na paddanoj slici
	const unsigned int i = blockIdx.y * blockDim.y + threadIdx.y + paddingSize;
	const unsigned int j = blockIdx.x * blockDim.x + threadIdx.x + paddingSize;

	// Osim kod kvadratnih slika (u ovakvom odabiru veli�ine bloka), nemogu�e je savr�eno pokriti sliku blokovima
	// Ova if naredba omogu�ava da se ra�un ne izvodi u onim threadovima koji su dio bloka koji ne pokriva dio slike
	// Tako�er osigurava da se ignorira okvir slike tj. padding

	if (j >= paddingSize && j < paddedWidth - paddingSize && i >= paddingSize && i < paddedHeight - paddingSize) {
		unsigned int outputPixelPosition = (i - paddingSize) * width + (j - paddingSize);
		result[outputPixelPosition] = 0.0;
		for (int k = -S; k <= S; k++) {
			for (int l = -S; l <= S; l++) {
				unsigned int inputPixelPosition = (i + k) * paddedWidth + (j + l);
				unsigned int coefPos = (k + S) * filterSize + (l + S);
				sum += filter[coefPos];
				result[outputPixelPosition] += devicePaddedImage[inputPixelPosition] * filter[coefPos];
			}
		}
		// Normalizacija
		if (result[outputPixelPosition] > 0.0) {
			result[outputPixelPosition] = result[outputPixelPosition] / sum;
		}
		else
			result[outputPixelPosition] = 0.0;
	}
};

int ZeroPadding(float* fmap, const unsigned int& width, const unsigned int& height,
	const int& filterSize,
	float* paddedImage, const unsigned int& paddedwidth, const unsigned int& paddedheight);

inline unsigned int DivideCieling(const unsigned int& a, const unsigned int& b);

int main()
{
	// Inicijalizacija vektora koji sadr�i sliku
	std::vector<unsigned char> image;
	unsigned int width, height;

	// Svaki filter se izvodi iz broja S, S je veli�ina paddinga
	// Padding je jedan od na�ina za popravak rubova konvolucije
	unsigned int S = 1;
	unsigned int filterSize = 2 * S + 1;

	// U�itavanje slike u formatu RGBARGBARGBA - sirovi podaci
	unsigned error = lodepng::decode(image, width, height, "slika.png");
	// Neuspjelo u�itavanje slike
	if (error) std::cout << "Decoder error -> " << error << ": " << std::endl;

	// Alokacija memorije za spremanje slike u formatu piksela kroz 3 kanala (RGB)
	// Kompletna slika je veli�ine width * height * 3 kanala
	float* inputImage = new float[(image.size() * 3) / 4];

	// Alokacija memorija za svakoi pojedini kanal
	float* inputImageRED = new float[image.size() / 4];
	float* inputImageGREEN = new float[image.size() / 4];
	float* inputImageBLUE = new float[image.size() / 4];

	float* outputImageR = new float[image.size() / 4];
	float* outputImageG = new float[image.size() / 4];
	float* outputImageB = new float[image.size() / 4];

	// pixelCount govori o kojem se trenutno pikselu radi u RGB nizu. Svako RGBA ponavljanje 
	// je jedan piksel
	int pixelCount = 0;

	// Ignorira se alpha vrijednost jer je bitna samo boja

	for (int i = 0; i < image.size(); i = i + 4) {
		// Pull each RGB pixel to its array
		inputImageRED[pixelCount] = (float)image.at(i);
		inputImageGREEN[pixelCount] = (float)image.at(i + 1);
		inputImageBLUE[pixelCount] = (float)image.at(i + 2);
		pixelCount++;
	}


	// Alokacija memorije za filter na hostu
	float* filter = new float[filterSize * filterSize];
	// Popunjavanje filtera vrijednostima
	filter[0] = 1; filter[1] = 2; filter[2] = 1;
	filter[3] = 2; filter[4] = 4; filter[5] = 2;
	filter[6] = 1; filter[7] = 2; filter[8] = 1;

	// Odabr filtera
	//LoadFilter(filter, filterSize, "GAUSS");

	//Ra�unanje paddinga 
	unsigned int paddedWidth = width + 2 * S;
	unsigned int paddedHeight = height + 2 * S;

	// Alokacija memorije za sliku pro�irenu paddingom
	float* paddedImageR = new float[paddedWidth * paddedHeight];
	float* paddedImageG = new float[paddedWidth * paddedHeight];
	float* paddedImageB = new float[paddedWidth * paddedHeight];

	// Dodavanje paddinga
	ZeroPadding(inputImageRED, width, height, S, paddedImageR, paddedWidth, paddedHeight);
	ZeroPadding(inputImageGREEN, width, height, S, paddedImageG, paddedWidth, paddedHeight);
	ZeroPadding(inputImageBLUE, width, height, S, paddedImageB, paddedWidth, paddedHeight);


	// Alokacija memorije za sliku na device-u i transfer paddane slike sa hosta na device
	float* devicePaddedImageR;
	float* devicePaddedImageG;
	float* devicePaddedImageB;

	unsigned int paddedImageSizeByte = paddedWidth * paddedHeight * sizeof(float);

	hipMalloc(reinterpret_cast<void**>(&devicePaddedImageR), paddedImageSizeByte);
	hipMemcpy(devicePaddedImageR, paddedImageR, paddedImageSizeByte, hipMemcpyHostToDevice);
	hipMalloc(reinterpret_cast<void**>(&devicePaddedImageG), paddedImageSizeByte);
	hipMemcpy(devicePaddedImageG, paddedImageG, paddedImageSizeByte, hipMemcpyHostToDevice);
	hipMalloc(reinterpret_cast<void**>(&devicePaddedImageB), paddedImageSizeByte);
	hipMemcpy(devicePaddedImageB, paddedImageB, paddedImageSizeByte, hipMemcpyHostToDevice);// Host to Device

	// Alokacija i transfer filtera na device
	float* deviceFilter;
	unsigned int filterKernelSizeByte = filterSize * filterSize * sizeof(float);
	hipMalloc(reinterpret_cast<void**>(&deviceFilter), filterKernelSizeByte);
	hipMemcpy(deviceFilter, filter, filterKernelSizeByte, hipMemcpyHostToDevice);


	// Postavljanje konfiguracije izvedbe
	// Koriste se blokovi veli�ine 16x16 da se osigura dovoljan broj threadova
	// koji �e izvr�avati zadatke. U ovom programu se ne zna kojoj grafi�koj kartici se izvodi 
	// program pa je 16x16 zlatna sredina. Bitno je da blok bude veli�ine
	// potencije broja 2
	const unsigned int blockWidth = 16;
	const unsigned int blockHeight = 16;

	// Funkcija DivideCeiling vra�a ve�u vrijednost prilikom cjelobrojnog dijeljenja
	// Potrebno je izra�unati grid tj. podijeliti ulaznu sliku na blokove, a te blokove
	// na threadove koji �e ra�unati konvoluciju na svakom pikselu slike
	// Veli�ina grida je width/blockWidth X height/blockHeight = 120 x 68 blokova
	// Veli�ina bloka je 16x16 threadova
	// Sve skupa 2088960 threadova izvr�avanja. Svaki piksel dobiva svoj thread izvr�avanja
	// Naravno, �to je bolja grafi�ka kartica bit �e mogu�e izvr�iti vi�e threadova
	// istovremeno.

	const dim3 grid(DivideCieling(width, blockWidth), DivideCieling(height, blockHeight));
	const dim3 threadBlock(blockWidth, blockHeight);

	// Alokacija i transfer memorije na GPU koja sadr�ava rezultat konvolucije
	// Bitno je konvoluciju raditi na kompletno novoj slici radi kazualnosti tj.
	// da nam ve� izra�unati pikseli ne utje�u na neizra�unate
	float* deviceResultR;
	float* deviceResultG;
	float* deviceResultB;
	unsigned int imageSizeByte = width * height * sizeof(float);
	
	hipMalloc(reinterpret_cast<void**>(&deviceResultR), imageSizeByte);
	hipMalloc(reinterpret_cast<void**>(&deviceResultG), imageSizeByte);
	hipMalloc(reinterpret_cast<void**>(&deviceResultB), imageSizeByte);

	// Poziv kernel funkcije
	ConvolutionKernel <<<grid, threadBlock >>> (devicePaddedImageR, paddedWidth, paddedHeight, deviceFilter, S, deviceResultR, width, height);
	ConvolutionKernel <<<grid, threadBlock >>> (devicePaddedImageG, paddedWidth, paddedHeight, deviceFilter, S, deviceResultG, width, height);
	ConvolutionKernel <<<grid, threadBlock >>> (devicePaddedImageB, paddedWidth, paddedHeight, deviceFilter, S, deviceResultB, width, height);
	// Kopiranje memorije natrag na host
	hipMemcpy(outputImageR, deviceResultR, imageSizeByte, hipMemcpyDeviceToHost); 
	hipMemcpy(outputImageG, deviceResultG, imageSizeByte, hipMemcpyDeviceToHost); 
	hipMemcpy(outputImageB, deviceResultB, imageSizeByte, hipMemcpyDeviceToHost); 

	// Bitno je �ekati da se svi izra�uni izvr�e i za to se koristi sinkronizacijska funkcija
	// U suprotnom, host tj. CPU bi zapo�eo dekodiranje slike za koju jo� nisu gotovi izra�uni jer
	// CPU i GPU mogu raditi odvojeno zadatke
	hipDeviceSynchronize();


	// Spajanje RGB kanala u jednu sliku. Dodaje se i alpha kanal zbog .png formata
	std::vector<unsigned char> outputImage;

	for (int i = 0; i < width; i++) {
		for (int j = 0; j < height; j++) {
			outputImage.push_back(outputImageR[i * height + j]);
			outputImage.push_back(outputImageG[i * height + j]);
			outputImage.push_back(outputImageB[i * height + j]);
			outputImage.push_back(255); 
		}
	}

	// Enkodiranje i spremanje slike na disk
	error = lodepng::encode("konvolucija.png", outputImage, width, height);
	if (error) std::cout << "encoder error " << error << ": " << std::endl;

	delete[] inputImage;
	return 0;
}


int ZeroPadding(float* inputImage, const unsigned int& width, const unsigned int& height, const int& filterSize,
	float* paddedImage, const unsigned int& paddedwidth, const unsigned int& paddedheight)
{

	if (paddedImage == NULL) printf("wtf?");

	for (unsigned int i = 0; i < paddedheight; i++) {
		for (unsigned int j = 0; j < paddedwidth; j++) {

			// Set the pixel position of the padded fmap
			unsigned int paddedPixelPos = i * paddedwidth + j;

			// Copy the pixel value
			if (i >= filterSize && i < height + filterSize &&
				j >= filterSize && j < width + filterSize) {
				unsigned int pixelPos = (i - filterSize) * width + (j - filterSize);
				paddedImage[paddedPixelPos] = inputImage[pixelPos];
			}
			else {
				paddedImage[paddedPixelPos] = 0;
			}
		}
	}

	return 0;
};

inline unsigned int DivideCieling(const unsigned int& a, const unsigned int& b) { return (a % b != 0) ? (a / b + 1) : (a / b); };

//int LoadFilter(float* filter, int filterSize, std::string filterName) {
//	if (filterName == "GAUSS") {
//		for (int i = 0; i < filterSize * filterSize; i++) {
//			for (int j = 0; j < filterSize * filterSize; j++)
//				*(filter + 2 * i + j) = gaussBlur[i][j];
//			return 0;
//		}
//	}
//}

